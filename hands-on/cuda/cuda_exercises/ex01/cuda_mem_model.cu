// includes, system

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include <chrono>

// Here you can set the device ID that was assigned to you
#define MYDEVICE 0

// Simple utility function to check for CUDA runtime errors
void checkCUDAError(const char *msg);

///////////////////////////////////////////////////////////////////////////////
// Program main
///////////////////////////////////////////////////////////////////////////////
int main( int argc, char** argv) 
{
    hipSetDevice(MYDEVICE);
    // pointer and dimension for host memory
    int n, dimA;
    float *h_a;

    // pointers for device memory
    float *d_a, *d_b;

    // allocate and initialize host memory
    // Bonus: try using cudaMallocHost in place of malloc
    dimA = 8;
    //h_a = (float *) malloc(dimA*sizeof(float));
    hipHostMalloc(&h_a, dimA*sizeof(float), hipHostMallocDefault);
    printf("Allocation done!\n");
    for (n=0; n<dimA; n++)
    {
        h_a[n] = (float) n;
    }

    // Part 1 of 5: allocate device memory
    size_t memSize = dimA*sizeof(float);
    hipMalloc(&d_a, memSize);
    hipMalloc(&d_b, memSize);

    // Part 2 of 5: host to device memory copy
    auto start = std::chrono::system_clock::now();
    hipMemcpy(d_a, h_a, memSize, hipMemcpyHostToDevice);
    auto end = std::chrono::system_clock::now();
    std::chrono::duration<double> dur= end - start;
    printf("Time spent in seconds: %f\n", dur.count());
    auto bandwidth = (memSize/dur.count())/1000000;
    printf("PCI Express Bandwidth in MB/s: %f\n", bandwidth);

    // Part 3 of 5: device to device memory copy
    hipMemcpy(d_b, d_a, memSize, hipMemcpyDeviceToDevice);

    // clear host memory
    for (n=0; n<dimA; n++)
    {
        h_a[n] = 0.f;
    }

    // Part 4 of 5: device to host copy
    hipMemcpy(h_a, d_b, memSize, hipMemcpyDeviceToHost);

    // Check for any CUDA errors
    checkCUDAError("hipMemcpy calls");

    // verify the data on the host is correct
    for (n=0; n<dimA; n++)
    {
        assert(h_a[n] == (float) n);
    }

    // Part 5 of 5: free device memory pointers d_a and d_b
    hipFree(d_a);
    hipFree(d_b);

    // Check for any CUDA errors
    checkCUDAError("hipFree");

    // free host memory pointer h_a
    //free(h_a);
    hipFree(h_a);
    // If the program makes it this far, then the results are correct and
    // there are no run-time errors.  Good work!
    printf("Correct!\n");

    return 0;
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(-1);
    }                         
}
